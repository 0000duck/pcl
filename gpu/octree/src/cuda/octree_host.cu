#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
*
*  Copyright (c) 2011, Willow Garage, Inc.
*  All rights reserved.
*
*  Redistribution and use in source and binary forms, with or without
*  modification, are permitted provided that the following conditions
*  are met:
*
*   * Redistributions of source code must retain the above copyright
*     notice, this list of conditions and the following disclaimer.
*   * Redistributions in binary form must reproduce the above
*     copyright notice, this list of conditions and the following
*     disclaimer in the documentation and/or other materials provided
*     with the distribution.
*   * Neither the name of Willow Garage, Inc. nor the names of its
*     contributors may be used to endorse or promote products derived
*     from this software without specific prior written permission.
*
*  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
*  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
*  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
*  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
*  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
*  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
*  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
*  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
*  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
*  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
*  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
*  POSSIBILITY OF SUCH DAMAGE.
*
*  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
*/

#include "pcl/gpu/utils/timers_cuda.hpp"
#include "pcl/gpu/utils/safe_call.hpp"

#include "internal.hpp"
#include "utils/boxutils.hpp"

#include <assert.h>
#include<algorithm>
#include<limits>

using namespace pcl::gpu;
using namespace pcl::device;

namespace pcl
{
    namespace device
    {
        __global__ void get_cc_kernel(int *data)
        {
            data[threadIdx.x + blockDim.x * blockIdx.x] = threadIdx.x;
        }

        struct ChildNode
        {
            uint3 index;
            std::uint8_t mask_pos;

            ChildNode(){}

            ChildNode(uint3 i, uint8_t mp)
            {
                index = i;
                mask_pos = mp;
            }
        };
    }
}

void  pcl::device::OctreeImpl::get_gpu_arch_compiled_for(int& bin, int& ptx)
{
    hipFuncAttributes attrs;
    cudaSafeCall( hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(get_cc_kernel)) );  
    bin = attrs.binaryVersion;
    ptx = attrs.ptxVersion;
}

void pcl::device::OctreeImpl::setCloud(const PointCloud& input_points)
{
    points = input_points;
}

void pcl::device::OctreeImpl::internalDownload()
{
    int number;
    DeviceArray<int>(octreeGlobal.nodes_num, 1).download(&number); 

    DeviceArray<int>(octreeGlobal.begs,  number).download(host_octree.begs);    
    DeviceArray<int>(octreeGlobal.ends,  number).download(host_octree.ends);    
    DeviceArray<int>(octreeGlobal.nodes, number).download(host_octree.nodes);    
    DeviceArray<int>(octreeGlobal.codes, number).download(host_octree.codes); 

    points_sorted.download(host_octree.points_sorted, host_octree.points_sorted_step);    
    indices.download(host_octree.indices);    

    host_octree.downloaded = true;
}

namespace 
{
    int getBitsNum(int integer)
    {
        int count = 0;
        while(integer > 0)
        {
            if (integer & 1)
                ++count;
            integer>>=1;
        }
        return count;
    } 

    ChildNode nearestVoxel(const OctreeImpl::PointType& query, const unsigned& level, const std::uint8_t& mask, const float3& minp, const float3& maxp, const uint3& index)
    {
        assert(mask != 0);
        //identify closest voxel
        float closest_distance = std::numeric_limits<float>::max();
        unsigned closest_index = 0;
        uint3 closest = make_uint3(0,0,0);
        const unsigned voxel_width = 1 << (level + 2);

        for (unsigned i = 0; i < 8; ++i)
        {
            if ((mask & (1<<i)) == 0)   //no child
                continue;

            uint3 child;
            child.x = (index.x << 1) + (i & 1);
            child.y = (index.y << 1) + ((i>>1) & 1);
            child.z = (index.z << 1) + ((i>>2) & 1);

            //find center of child cell
            float3 voxel_center;
            voxel_center.x = minp.x + (maxp.x - minp.x) * (2*child.x + 1) / voxel_width;
            voxel_center.y = minp.y + (maxp.y - minp.y) * (2*child.y + 1) / voxel_width;
            voxel_center.z = minp.z + (maxp.z - minp.z) * (2*child.z + 1) / voxel_width;

            //compute distance to centroid
            const float3 dist = make_float3((voxel_center.x - query.x), (voxel_center.y - query.y), (voxel_center.z - query.z));

            float distance_to_query = dist.x * dist.x + dist.y * dist.y + dist.z * dist.z;

            //compare distance
            if (distance_to_query < closest_distance)
            {
                closest_distance = distance_to_query;
                closest_index = i;
                closest.x = child.x;
                closest.y = child.y;
                closest.z = child.z;
            }
        }

        return ChildNode(make_uint3(closest.x, closest.y, closest.z), (1<<closest_index));
    }

    struct OctreeIteratorHost
    {        
        const static int MAX_LEVELS_PLUS_ROOT = 11;
        int paths[MAX_LEVELS_PLUS_ROOT];          
        int level;

        OctreeIteratorHost()
        {
            level = 0; // root level
            paths[level] = (0 << 8) + 1;                    
        }

        void gotoNextLevel(int first, int len) 
        {   
            ++level;
            paths[level] = (first << 8) + len;        
        }       

        int operator*() const 
        { 
            return paths[level] >> 8; 
        }        

        void operator++()
        {
            while(level >= 0)
            {
                int data = paths[level];

                if ((data & 0xFF) > 1) // there are another siblings, can goto there
                {                           
                    data += (1 << 8) - 1;  // +1 to first and -1 from len
                    paths[level] = data;
                    break;
                }
                else
                    --level; //goto parent;            
            }        
        }        
    };

}

void pcl::device::OctreeImpl::radiusSearchHost(const PointType& query, float radius, std::vector<int>& out, int max_nn) const
{            
    out.clear();  

    float3 center = make_float3(query.x, query.y, query.z);

    OctreeIteratorHost iterator;

    while(iterator.level >= 0)
    {        
        int node_idx = *iterator;
        int code = host_octree.codes[node_idx];

        float3 node_minp = octreeGlobal.minp;
        float3 node_maxp = octreeGlobal.maxp;        
        calcBoundingBox(iterator.level, code, node_minp, node_maxp);

        //if true, take nothing, and go to next
        if (checkIfNodeOutsideSphere(node_minp, node_maxp, center, radius))        
        {                
            ++iterator;            
            continue;
        }

        //if true, take all, and go to next
        if (checkIfNodeInsideSphere(node_minp, node_maxp, center, radius))
        {            
            int beg = host_octree.begs[node_idx];
            int end = host_octree.ends[node_idx];

            end = beg + std::min<int>((int)out.size() + end - beg, max_nn) - (int)out.size();

            out.insert(out.end(), host_octree.indices.begin() + beg, host_octree.indices.begin() + end);
            if (out.size() == (std::size_t)max_nn)
                return;

            ++iterator;
            continue;
        }

        // test children
        int children_mask = host_octree.nodes[node_idx] & 0xFF;

        bool isLeaf = children_mask == 0;

        if (isLeaf)
        {            
            const int beg = host_octree.begs[node_idx];
            const int end = host_octree.ends[node_idx];                                    

            for(int j = beg; j < end; ++j)
            {
                int index = host_octree.indices[j];
                float point_x = host_octree.points_sorted[j                                     ];
                float point_y = host_octree.points_sorted[j + host_octree.points_sorted_step    ];
                float point_z = host_octree.points_sorted[j + host_octree.points_sorted_step * 2];

                float dx = (point_x - center.x);
                float dy = (point_y - center.y);
                float dz = (point_z - center.z);

                float dist2 = dx * dx + dy * dy + dz * dz;

                if (dist2 < radius * radius)
                    out.push_back(index);

                if (out.size() == (std::size_t)max_nn)
                    return;
            }               
            ++iterator;               
            continue;
        }

        int first  = host_octree.nodes[node_idx] >> 8;        
        iterator.gotoNextLevel(first, getBitsNum(children_mask));                
    }
}

void  pcl::device::OctreeImpl::approxNearestSearchHost(const PointType& query, int& out_index, float& sqr_dist) const
{
    const float3& minp = octreeGlobal.minp;
    const float3& maxp = octreeGlobal.maxp;

    size_t node_idx = 0;
    const auto code = CalcMorton(minp, maxp)(query);
    unsigned level = 0;

    bool voxel_traversal = false;
    uint3 full_index = Morton::decomposeCode(code);

    while(true)
    {
        const auto node = host_octree.nodes[node_idx];
        const std::uint8_t mask = node & 0xFF;

        if(!mask)  // leaf
            break;

        ChildNode child_node;
        if (!voxel_traversal)    // no empty voxel encountered yet, performing morton code based traversal
        {
            child_node.mask_pos = 1 << Morton::extractLevelCode(code, level);

            if (!(mask & child_node.mask_pos)) // child doesn't exist
            {
                full_index.x >>= (Morton::levels - level);
                full_index.y >>= (Morton::levels - level);
                full_index.z >>= (Morton::levels - level);

                voxel_traversal = true;  //switch to nearest-centroid based traversal
                child_node = nearestVoxel(query, level, mask, minp, maxp, full_index);
            }
        }
        else
        child_node = nearestVoxel(query, level, mask, minp, maxp, child_node.index);

        node_idx = (node >> 8) + getBitsNum(mask & (child_node.mask_pos - 1));
        ++level;
    }

    int beg = host_octree.begs[node_idx];
    int end = host_octree.ends[node_idx];

    sqr_dist = std::numeric_limits<float>::max();

    for(int i = beg; i < end; ++i)
    {
        float point_x = host_octree.points_sorted[i                                     ];
        float point_y = host_octree.points_sorted[i + host_octree.points_sorted_step    ];
        float point_z = host_octree.points_sorted[i + host_octree.points_sorted_step * 2];

        float dx = (point_x - query.x);
        float dy = (point_y - query.y);
        float dz = (point_z - query.z);

        float d2 = dx * dx + dy * dy + dz * dz;

        if (sqr_dist > d2)
        {
            sqr_dist = d2;
            out_index = i;
        }
    }

    out_index = host_octree.indices[out_index];
}
