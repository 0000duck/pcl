#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Copyright (c) 2011, Willow Garage, Inc.
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 */

#include <limits>

#include "internal.hpp"
#include "pcl/gpu/utils/device/warp.hpp"

#include "utils/copygen.hpp"
#include "utils/boxutils.hpp"
#include "utils/scan_block.hpp"


namespace pcl { namespace device { namespace appnearest_search
{   
    using PointType = OctreeImpl::PointType;
	
	struct Batch
	{   
		const PointType* queries;

		const int *indices;
		const float* points;
		int points_step; // elem step

		OctreeGlobalWithBox octree;
	    
		int queries_num;                
		mutable int* output;                
	};

	struct KernelPolicy
	{
		enum 
		{
			CTA_SIZE = 512,

			LOG_WARP_SIZE = 5,
			WARP_SIZE = 1 << LOG_WARP_SIZE,
			WARPS_COUNT = CTA_SIZE/WARP_SIZE,                    
		};	
	};

	struct Warp_appNearestSearch
	{   
	public:                		
		const Batch& batch;

		int query_index;        
		float3 query;  
		int result_idx;

		__device__ __forceinline__ Warp_appNearestSearch(const Batch& batch_arg, int query_index_arg) 
			: batch(batch_arg), query_index(query_index_arg){}

		__device__ __forceinline__ void launch(bool active)
		{              
			int node_idx = -1;
			if (active)
			{
				PointType q = batch.queries[query_index];
				query = make_float3(q.x, q.y, q.z);                        

				node_idx = findNode();
			}           

			processNode(node_idx);                    

			if (active)
				batch.output[query_index] = batch.indices[result_idx];
		}    

	private:

		__device__ __forceinline__ int findNode()
		{
			float3 minp = batch.octree.minp;
			float3 maxp = batch.octree.maxp;

			int node_idx = 0;
			int code = CalcMorton(minp, maxp)(query);
			int level = 0;

			bool centroid_traversal = false;
			int mask_pos;
			int x, y, z;

			for(;;)
			{
				int node = batch.octree.nodes[node_idx];
				int mask = node & 0xFF;

				float3 query_point;
				query_point.x = query.x;
				query_point.y = query.y;
				query_point.z = query.z;

				if(__popc(mask) == 0)  // leaf
				{
					//printf ("node x %d\n", node_idx);
					return node_idx;
				}

				if (!centroid_traversal)    // no empty voxel encountered yet, performing morton code based traversal
				{
					mask_pos = 1 << Morton::extractLevelCode(code, level);

					if ( (mask & mask_pos) == 0) // no child
					{

						//find current cell
						Morton::decomposeCode(code, x, y, z);

						x >>= (Morton::levels - level);
						y >>= (Morton::levels - level);
						z >>= (Morton::levels - level);

						centroid_traversal = true;  //switch to nearest-centroid based traversal
						mask_pos = nearestVoxelTraversal(query_point, level, mask, minp, maxp, x, y, z);
					}
				}

				else
					mask_pos = nearestVoxelTraversal(query_point, level, mask, minp, maxp, x, y, z);

				node_idx = (node >> 8) + __popc(mask & (mask_pos - 1));
				++level;
			}
		};

		__device__ int nearestVoxelTraversal(float3 query, int level, int mask, float3 minp, float3 maxp, int& x, int& y, int& z)
		{
			//identify closest voxel
			float closest_distance = std::numeric_limits<float>::max();
			int closest_index = 0, closest_x = 0, closest_y = 0, closest_z = 0;

			for (int i = 0; i < 8; ++i)
			{
				if ((mask & (1<<i)) == 0)   //no child
					continue;

				//calculate  x,y,z offset for voxel
				int x_cord = i & 1;
				int y_cord = (i>>1) & 1;
				int z_cord = (i>>2) & 1;

				int x_child, y_child, z_child;
				x_child = x*2 + x_cord;
				y_child = y*2 + y_cord;
				z_child = z*2 + z_cord;

				//find center of child cell
				float3 voxel_center;
				voxel_center.x = minp.x + (maxp.x - minp.x) * (2*x_child + 1) / (2 * 1<<(level + 1));
				voxel_center.y = minp.y + (maxp.y - minp.y) * (2*y_child + 1) / (2 * 1<<(level + 1));
				voxel_center.z = minp.z + (maxp.z - minp.z) * (2*z_child + 1) / (2 * 1<<(level + 1));

				//compute distance to centroid
				float dx = (voxel_center.x - query.x);
				float dy = (voxel_center.y - query.y);
				float dz = (voxel_center.z - query.z);
				float distance_to_query = dx * dx + dy * dy + dz * dz;

				//compare distance
				if (distance_to_query < closest_distance)
				{
					closest_distance = distance_to_query;
					closest_index = i;
					closest_x = x_child;
					closest_y = y_child;
					closest_z = z_child;
				}
			}

			x = closest_x;
			y = closest_y;
			z = closest_z;

			return  (1<<closest_index);
		}

		__device__ __forceinline__ void processNode(int node_idx)
		{   
            __shared__ volatile int  per_warp_buffer[KernelPolicy::WARPS_COUNT];

			int mask = __ballot_sync(0xFFFFFFFF, node_idx != -1);                        

			while(mask)
			{                
				unsigned int laneId = Warp::laneId();
				unsigned int warpId = threadIdx.x/warpSize;            

				int active_lane = __ffs(mask) - 1; //[0..31]                        
				mask &= ~(1 << active_lane);   

				volatile int* warp_buffer = &per_warp_buffer[warpId];

				//broadcast beg
				if (active_lane == laneId)
					*warp_buffer = batch.octree.begs[node_idx];                    
				int beg = *warp_buffer;

				//broadcast end
				if (active_lane == laneId)
					*warp_buffer = batch.octree.ends[node_idx];
				int end = *warp_buffer;

				float3 active_query;
				volatile float* warp_buffer_float = (float*)&per_warp_buffer[warpId];

				//broadcast warp_query
				if (active_lane == laneId)
					*warp_buffer_float = query.x;
				active_query.x = *warp_buffer_float;

				if (active_lane == laneId)
					*warp_buffer_float = query.y;
				active_query.y = *warp_buffer_float;

				if (active_lane == laneId)
					*warp_buffer_float = query.z;
				active_query.z = *warp_buffer_float;                            

				int offset = NearestWarpKernel<KernelPolicy::CTA_SIZE>(beg, batch.points_step, end - beg, active_query);                    

				if (active_lane == laneId)
					result_idx = beg + offset;
			}
		}

        template<int CTA_SIZE>
		__device__ __forceinline__ int NearestWarpKernel(int beg, int points_step, int length, const float3& active_query)
		{                        						
            __shared__ volatile float dist2[CTA_SIZE];
            __shared__ volatile int   index[CTA_SIZE];
			
            int tid = threadIdx.x;
			dist2[tid] = std::numeric_limits<float>::max();

			//serial step
            for (int idx = Warp::laneId(); idx < length; idx += Warp::STRIDE)
			{
				float dx = batch.points[beg + idx                  ] - active_query.x;
				float dy = batch.points[beg + idx + points_step    ] - active_query.y;
				float dz = batch.points[beg + idx + points_step * 2] - active_query.z;

				float d2 = dx * dx + dy * dy + dz * dz;

				if (dist2[tid] > d2)
				{
					dist2[tid] = d2;
					index[tid] = idx;                            
				}
			}
			//parallel step
			unsigned int lane = Warp::laneId();

			float mind2 = dist2[tid];

			if (lane < 16)
			{
				float next = dist2[tid + 16];
				if (mind2 > next) 
				{ 
					dist2[tid] = mind2 = next; 
					index[tid] = index[tid + 16]; 
				}                        
			}

			if (lane < 8)
			{
				float next = dist2[tid + 8];
				if (mind2 > next) 
				{ 
					dist2[tid] = mind2 = next; 
					index[tid] = index[tid + 8]; 
				}                        
			}

			if (lane < 4)
			{
				float next = dist2[tid + 4];
				if (mind2 > next) 
				{ 
					dist2[tid] = mind2 = next; 
					index[tid] = index[tid + 4]; 
				}                        
			}

			if (lane < 2)
			{
				float next = dist2[tid + 2];
				if (mind2 > next) 
				{ 
					dist2[tid] = mind2 = next; 
					index[tid] = index[tid + 2]; 
				}                        
			}

			if (lane < 1)
			{
				float next = dist2[tid + 1];
				if (mind2 > next) 
				{ 
					dist2[tid] = mind2 = next; 
					index[tid] = index[tid + 1]; 
				}                        
			}        

			return index[tid - lane];
		}
	};
	
	__global__ void KernelAN(const Batch batch) 
	{         
		int query_index = blockIdx.x * blockDim.x + threadIdx.x;

		bool active = query_index < batch.queries_num;

		if (__all_sync(0xFFFFFFFF, active == false)) 
			return;

		Warp_appNearestSearch search(batch, query_index);
		search.launch(active); 
	}

} } }


void pcl::device::OctreeImpl::approxNearestSearch(const Queries& queries, NeighborIndices& results) const
{
    using BatchType = pcl::device::appnearest_search::Batch;

    BatchType batch;
    batch.indices = indices;
    batch.octree = octreeGlobal;

    batch.queries_num = (int)queries.size();        
    batch.output = results.data;     

    batch.points = points_sorted;
    batch.points_step = (int)points_sorted.elem_step();
    batch.queries = queries;

    int block = pcl::device::appnearest_search::KernelPolicy::CTA_SIZE;
    int grid = (batch.queries_num + block - 1) / block;    

    cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(pcl::device::appnearest_search::KernelAN), hipFuncCachePreferL1) );

    pcl::device::appnearest_search::KernelAN<<<grid, block>>>(batch);
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );
}